#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#include <sisci_error.h>
#include <sisci_api.h>

#include "c63.h"
#include "c63_write.h"
#include "common.h"
#include "tables.h"
#include <time.h>

static char *output_file, *input_file;
FILE *outfile;

static uint32_t remote_node = 0;
static int limit_numframes = 0;

static uint32_t width;
static uint32_t height;

/* getopt */
extern int optind;
extern char *optarg;

/* Read planar YUV frames with 4:2:0 chroma sub-sampling */
static yuv_t *read_yuv(FILE *file, struct c63_common *cm)
{
    size_t len = 0;
    yuv_t *image = (yuv_t *)malloc(sizeof(*image));

    /* Read Y. The size of Y is the same as the size of the image. */
    image->Y = (uint8_t *)calloc(1, cm->padw[Y_COMPONENT] * cm->padh[Y_COMPONENT]);
    len += fread(image->Y, 1, width * height, file);

    /* Read U. Given 4:2:0 chroma sub-sampling, the size is 1/4 of Y */
    image->U = (uint8_t *)calloc(1, cm->padw[U_COMPONENT] * cm->padh[U_COMPONENT]);
    len += fread(image->U, 1, (width * height) / 4, file);

    /* Read V. Given 4:2:0 chroma sub-sampling, the size is 1/4 of Y. */
    image->V = (uint8_t *)calloc(1, cm->padw[V_COMPONENT] * cm->padh[V_COMPONENT]);
    len += fread(image->V, 1, (width * height) / 4, file);

    if (ferror(file))
    {
        perror("ferror");
        exit(EXIT_FAILURE);
    }

    if (feof(file))
    {
        free(image->Y);
        free(image->U);
        free(image->V);
        free(image);
        return NULL;
    }
    else if (len != width * height * 1.5)
    {
        fprintf(stderr, "Reached end of file, but incorrect bytes read.\n");
        fprintf(stderr, "Wrong input? (height: %d width: %d)\n", height, width);
        free(image->Y);
        free(image->U);
        free(image->V);
        free(image);
        return NULL;
    }

    return image;
}

struct c63_common *
init_c63_enc( int width, int height )
{
    int i;

    /* calloc() sets allocated memory to zero */
    c63_common *cm =
        ( c63_common * ) calloc( 1, sizeof( struct c63_common ) );

    cm->width = width;
    cm->height = height;

    cm->padw[Y_COMPONENT] = cm->ypw =( uint32_t ) ( ceil( width / 16.0f ) * 16 );
    cm->padh[Y_COMPONENT] = cm->yph =( uint32_t ) ( ceil( height / 16.0f ) * 16 );
    cm->padw[U_COMPONENT] = cm->upw =( uint32_t ) ( ceil( width * UX / ( YX * 8.0f ) ) * 8 );
    cm->padh[U_COMPONENT] = cm->uph =( uint32_t ) ( ceil( height * UY / ( YY * 8.0f ) ) * 8 );
    cm->padw[V_COMPONENT] = cm->vpw =( uint32_t ) ( ceil( width * VX / ( YX * 8.0f ) ) * 8 );
    cm->padh[V_COMPONENT] = cm->vph =( uint32_t ) ( ceil( height * VY / ( YY * 8.0f ) ) * 8 );

    cm->mb_cols = cm->ypw / 8;
    cm->mb_rows = cm->yph / 8;

    /* Quality parameters -- Home exam deliveries should have original values,
       i.e., quantization factor should be 25, search range should be 16, and the
       keyframe interval should be 100. */
    cm->qp = 25;                // Constant quantization factor. Range: [1..50]
    cm->me_search_range = 16;   // Pixels in every direction
    cm->keyframe_interval = 100;        // Distance between keyframes

    /* Initialize quantization tables */
    for ( i = 0; i < 64; ++i )
    {
        cm->quanttbl[Y_COMPONENT][i] = yquanttbl_def[i] / ( cm->qp / 10.0 );
        cm->quanttbl[U_COMPONENT][i] = uvquanttbl_def[i] / ( cm->qp / 10.0 );
        cm->quanttbl[V_COMPONENT][i] = uvquanttbl_def[i] / ( cm->qp / 10.0 );
    }

    return cm;
}

void
free_c63_enc( struct c63_common *cm )
{
    destroy_frame( cm->curframe );
    free( cm );
}

// Main loop for client - Handles transfers and acknowledgements and writing
// Add these lines at the top after the existing includes in paste.txt

// Timing structures and functions for benchmarking
struct frame_timing {
    struct timespec frame_start, yuv_start, yuv_end, encode_start, encode_end, result_start, result_end, frame_end;
    long yuv_us, encode_us, result_us, total_us;
};

struct benchmark_stats {
    long frames, total_yuv_us, total_encode_us, total_result_us, total_roundtrip_us;
    long min_yuv_us, max_yuv_us, min_encode_us, max_encode_us, min_result_us, max_result_us, min_total_us, max_total_us;
};

static inline void get_time(struct timespec *ts) { clock_gettime(CLOCK_MONOTONIC, ts); }
static inline long time_diff_us(struct timespec *start, struct timespec *end) {
    return (end->tv_sec - start->tv_sec) * 1000000L + (end->tv_nsec - start->tv_nsec) / 1000L;
}

// Helper function to format time with appropriate units
void format_time(long us, char *buffer, size_t size) {
    if (us < 1000) {
        snprintf(buffer, size, "%ld μs", us);
    } else if (us < 1000000) {
        snprintf(buffer, size, "%.1f ms", us / 1000.0);
    } else {
        snprintf(buffer, size, "%.2f s", us / 1000000.0);
    }
}

void update_benchmark_stats(struct benchmark_stats *stats, struct frame_timing *timing) {
    stats->frames++;
    stats->total_yuv_us += timing->yuv_us;
    stats->total_encode_us += timing->encode_us;
    stats->total_result_us += timing->result_us;
    stats->total_roundtrip_us += timing->total_us;
    
    if (stats->frames == 1) {
        stats->min_yuv_us = stats->max_yuv_us = timing->yuv_us;
        stats->min_encode_us = stats->max_encode_us = timing->encode_us;
        stats->min_result_us = stats->max_result_us = timing->result_us;
        stats->min_total_us = stats->max_total_us = timing->total_us;
    } else {
        if (timing->yuv_us < stats->min_yuv_us) stats->min_yuv_us = timing->yuv_us;
        if (timing->yuv_us > stats->max_yuv_us) stats->max_yuv_us = timing->yuv_us;
        if (timing->encode_us < stats->min_encode_us) stats->min_encode_us = timing->encode_us;
        if (timing->encode_us > stats->max_encode_us) stats->max_encode_us = timing->encode_us;
        if (timing->result_us < stats->min_result_us) stats->min_result_us = timing->result_us;
        if (timing->result_us > stats->max_result_us) stats->max_result_us = timing->result_us;
        if (timing->total_us < stats->min_total_us) stats->min_total_us = timing->total_us;
        if (timing->total_us > stats->max_total_us) stats->max_total_us = timing->total_us;
    }
}

void print_benchmark_results(struct benchmark_stats *stats, uint32_t width, uint32_t height) {
    if (stats->frames == 0) return;
    
    char avg_yuv[32], min_yuv[32], max_yuv[32];
    char avg_encode[32], min_encode[32], max_encode[32];
    char avg_result[32], min_result[32], max_result[32];
    char avg_total[32], min_total[32], max_total[32];
    
    format_time(stats->total_yuv_us / stats->frames, avg_yuv, sizeof(avg_yuv));
    format_time(stats->min_yuv_us, min_yuv, sizeof(min_yuv));
    format_time(stats->max_yuv_us, max_yuv, sizeof(max_yuv));
    
    format_time(stats->total_encode_us / stats->frames, avg_encode, sizeof(avg_encode));
    format_time(stats->min_encode_us, min_encode, sizeof(min_encode));
    format_time(stats->max_encode_us, max_encode, sizeof(max_encode));
    
    format_time(stats->total_result_us / stats->frames, avg_result, sizeof(avg_result));
    format_time(stats->min_result_us, min_result, sizeof(min_result));
    format_time(stats->max_result_us, max_result, sizeof(max_result));
    
    format_time(stats->total_roundtrip_us / stats->frames, avg_total, sizeof(avg_total));
    format_time(stats->min_total_us, min_total, sizeof(min_total));
    format_time(stats->max_total_us, max_total, sizeof(max_total));
    
    printf("\n═══ BENCHMARK RESULTS (%ld frames) ═══\n", stats->frames);
    printf("YUV Transfer:    Avg:%8s  Min:%8s  Max:%8s\n", avg_yuv, min_yuv, max_yuv);
    printf("Encoding:        Avg:%8s  Min:%8s  Max:%8s\n", avg_encode, min_encode, max_encode);
    printf("Result Transfer: Avg:%8s  Min:%8s  Max:%8s\n", avg_result, min_result, max_result);
    printf("Total Roundtrip: Avg:%8s  Min:%8s  Max:%8s\n", avg_total, min_total, max_total);
    
    double yuv_mbps = (width * height * 1.5 * 8.0) / (stats->total_yuv_us / stats->frames);
    double fps = 1000000.0 / (stats->total_roundtrip_us / stats->frames);
    printf("YUV Transfer Rate: %.1f Mbps\n", yuv_mbps);
    printf("Processing Rate:   %.1f FPS\n", fps);
    printf("═══════════════════════════════════════\n");
}

// REPLACE the entire main_client_loop function with this version:
int main_client_loop(struct c63_common *cm, FILE *infile, int limit_numframes,
                    volatile struct client_segment *local_seg,
                    volatile struct server_segment *remote_seg,
                    sci_dma_queue_t dma_queue,
                    sci_local_segment_t local_segment,
                    sci_remote_segment_t remote_segment) 
{
    yuv_t *image;
    int numframes = 0;
    sci_error_t error;
    struct benchmark_stats stats = {0};
    struct frame_timing timing;
    
    printf("Client: Starting video encoding with benchmarking\n");
    
    // Dimensions exchange (unchanged)
    struct dimensions_data dim_data;
    dim_data.width = width;
    dim_data.height = height;
    
    memcpy((void*)local_seg->message_buffer, &dim_data, sizeof(struct dimensions_data));
    
    SCIStartDmaTransfer(dma_queue, local_segment, remote_segment,
        offsetof(struct client_segment, message_buffer), sizeof(struct dimensions_data),
        offsetof(struct server_segment, message_buffer), NO_CALLBACK, NULL, NO_FLAGS, &error);
                       
    if (error != SCI_ERR_OK) {
        fprintf(stderr, "Client: SCIStartDmaTransfer for dimensions failed - Error code 0x%x\n", error);
        return -1;
    }
    
    SCIWaitForDMAQueue(dma_queue, SCI_INFINITE_TIMEOUT, NO_FLAGS, &error);
    SCIFlush(NULL, NO_FLAGS);
    remote_seg->packet.cmd = CMD_DIMENSIONS;
    SCIFlush(NULL, NO_FLAGS);
    
    printf("Client: Waiting for server to acknowledge dimensions\n");
    time_t dim_start = time(NULL);
    bool dim_timeout = false;
    
    while (local_seg->packet.cmd != CMD_DIMENSIONS_ACK && !dim_timeout) {
        if (time(NULL) - dim_start > 30) {
            dim_timeout = true;
            fprintf(stderr, "Client: Timeout waiting for dimensions acknowledgment\n");
        }
    }
    
    if (dim_timeout) {
        fprintf(stderr, "Client: Failed to receive dimensions acknowledgment, exiting\n");
        return -1;
    }
    
    printf("Client: Dimensions acknowledged by server\n");
    local_seg->packet.cmd = CMD_INVALID;
    
    // Main processing loop with benchmarking
    while (1) {
        get_time(&timing.frame_start);
        
        image = read_yuv(infile, cm);
        if (!image) {
            printf("Client: End of input file reached\n");
            break;
        }

        printf("Processing frame %d, ", numframes);

        size_t y_size = width * height;
        size_t u_size = (width * height) / 4;
        size_t v_size = (width * height) / 4;
        size_t total_yuv_size = y_size + u_size + v_size;

        if (total_yuv_size > MESSAGE_SIZE) {
            fprintf(stderr, "Client: ERROR - Total YUV frame size (%zu) exceeds message buffer size (%d)\n", total_yuv_size, MESSAGE_SIZE);
            free(image->Y); free(image->U); free(image->V); free(image);
            return -1;
        }

        // Pack YUV data
        memcpy((void*)local_seg->message_buffer, image->Y, y_size);
        memcpy((void*)(local_seg->message_buffer + y_size), image->U, u_size);
        memcpy((void*)(local_seg->message_buffer + y_size + u_size), image->V, v_size);

        // Time YUV transfer
        get_time(&timing.yuv_start);
        SCIStartDmaTransfer(dma_queue, local_segment, remote_segment,
                        offsetof(struct client_segment, message_buffer), total_yuv_size,
                        offsetof(struct server_segment, message_buffer), NO_CALLBACK, NULL, NO_FLAGS, &error);

        if (error != SCI_ERR_OK) {
            fprintf(stderr, "Client: YUV frame DMA transfer failed - Error code 0x%x\n", error);
            free(image->Y); free(image->U); free(image->V); free(image);
            continue;
        }

        SCIWaitForDMAQueue(dma_queue, SCI_INFINITE_TIMEOUT, NO_FLAGS, &error);
        get_time(&timing.yuv_end);

        SCIFlush(NULL, NO_FLAGS);
        remote_seg->packet.cmd = CMD_YUV_DATA;
        remote_seg->packet.data_size = total_yuv_size;
        remote_seg->packet.y_size = y_size;
        remote_seg->packet.u_size = u_size;
        remote_seg->packet.v_size = v_size;
        SCIFlush(NULL, NO_FLAGS);

        // Wait for frame acknowledgment
        time_t frame_start = time(NULL);
        bool frame_timeout = false;
        while (local_seg->packet.cmd != CMD_YUV_DATA_ACK && !frame_timeout) {
            if (time(NULL) - frame_start > 30) {
                frame_timeout = true;
                fprintf(stderr, "Client: Timeout waiting for YUV frame acknowledgment\n");
            }
        }

        if (frame_timeout) {
            free(image->Y); free(image->U); free(image->V); free(image);
            continue;
        }

        local_seg->packet.cmd = CMD_INVALID;
        free(image->Y); free(image->U); free(image->V); free(image);

        // Mark encoding start (server is now processing)
        get_time(&timing.encode_start);

        // Wait for encoded data
        time_t encode_start = time(NULL);
        bool encode_timeout = false;
        
        while (local_seg->packet.cmd != CMD_ENCODED_DATA && !encode_timeout) {
            if (time(NULL) - encode_start > 120) {
                encode_timeout = true;
                fprintf(stderr, "Client: Timeout waiting for encoded data\n");
            }
        }
        
        if (encode_timeout) continue;

        // Mark encoding end and result transfer start
        get_time(&timing.encode_end);
        get_time(&timing.result_start);

        // Process encoded data (existing logic)
        size_t data_size = local_seg->packet.data_size;
        int keyframe = *((int*)local_seg->message_buffer);
        cm->curframe->keyframe = keyframe;
        
        char* encoded_data = (char*)local_seg->message_buffer + sizeof(int);
        
        size_t ydct_size = cm->ypw * cm->yph * sizeof(int16_t);
        memcpy(cm->curframe->residuals->Ydct, encoded_data, ydct_size);
        encoded_data += ydct_size;
        
        size_t udct_size = cm->upw * cm->uph * sizeof(int16_t);
        memcpy(cm->curframe->residuals->Udct, encoded_data, udct_size);
        encoded_data += udct_size;
        
        size_t vdct_size = cm->vpw * cm->vph * sizeof(int16_t);
        memcpy(cm->curframe->residuals->Vdct, encoded_data, vdct_size);
        encoded_data += vdct_size;
        
        size_t mby_size = cm->mb_rows * cm->mb_cols * sizeof(struct macroblock);
        memcpy(cm->curframe->mbs[Y_COMPONENT], encoded_data, mby_size);
        encoded_data += mby_size;
        
        size_t mbu_size = (cm->mb_rows/2) * (cm->mb_cols/2) * sizeof(struct macroblock);
        memcpy(cm->curframe->mbs[U_COMPONENT], encoded_data, mbu_size);
        encoded_data += mbu_size;
        
        size_t mbv_size = (cm->mb_rows/2) * (cm->mb_cols/2) * sizeof(struct macroblock);
        memcpy(cm->curframe->mbs[V_COMPONENT], encoded_data, mbv_size);

        get_time(&timing.result_end);

        // Acknowledge encoded data
        local_seg->packet.cmd = CMD_INVALID;
        SCIFlush(NULL, NO_FLAGS);
        remote_seg->packet.cmd = CMD_ENCODED_DATA_ACK;
        SCIFlush(NULL, NO_FLAGS);
        
        write_frame(cm);
        get_time(&timing.frame_end);
        
        // Calculate timings
        timing.yuv_us = time_diff_us(&timing.yuv_start, &timing.yuv_end);
        timing.encode_us = time_diff_us(&timing.encode_start, &timing.encode_end);
        timing.result_us = time_diff_us(&timing.result_start, &timing.result_end);
        timing.total_us = time_diff_us(&timing.frame_start, &timing.frame_end);
        
        update_benchmark_stats(&stats, &timing);
        
        char yuv_str[16], encode_str[16], result_str[16], total_str[16];
        format_time(timing.yuv_us, yuv_str, sizeof(yuv_str));
        format_time(timing.encode_us, encode_str, sizeof(encode_str));
        format_time(timing.result_us, result_str, sizeof(result_str));
        format_time(timing.total_us, total_str, sizeof(total_str));
        
        printf("YUV:%s, Encode:%s, Result:%s, Total:%s\n", 
               yuv_str, encode_str, result_str, total_str);
        
        cm->framenum++;
        cm->frames_since_keyframe++;
        if (cm->curframe->keyframe) {
            cm->frames_since_keyframe = 0;
        }
        
        ++numframes;
        
        if (limit_numframes && numframes >= limit_numframes) {
            printf("Client: Reached frame limit (%d frames), stopping\n", limit_numframes);
            break;
        }
    }
    
    // Print final results
    print_benchmark_results(&stats, width, height);
    
    SCIFlush(NULL, NO_FLAGS);
    remote_seg->packet.cmd = CMD_QUIT;
    SCIFlush(NULL, NO_FLAGS);
    
    printf("Client: Finished processing %d frames\n", numframes);
    return numframes;
}
static void print_help()
{
    printf("Usage: ./c63client -r nodeid [options] input_file\n");
    printf("Commandline options:\n");
    printf("  -r                             Node id of server\n");
    printf("  -h                             Height of images to compress\n");
    printf("  -w                             Width of images to compress\n");
    printf("  -o                             Output file (.c63)\n");
    printf("  [-f]                           Limit number of frames to encode\n");
    printf("\n");

    exit(EXIT_FAILURE);
}

int main(int argc, char **argv)
{
    unsigned int localAdapterNo = 0;
    int c;
    sci_error_t error;
    
    if (argc == 1) {
        print_help();
    }

    while ((c = getopt(argc, argv, "r:h:w:o:f:i:")) != -1)
    {
        switch (c)
        {
            case 'r':
                remote_node = atoi(optarg);
                break;
            case 'h':
                height = atoi(optarg);
                break;
            case 'w':
                width = atoi(optarg);
                break;
            case 'o':
                output_file = optarg;
                break;
            case 'f':
                limit_numframes = atoi(optarg);
                break;
            default:
                print_help();
                break;
        }
    }

    if (optind >= argc)
    {
        fprintf(stderr, "Error getting program options, try --help.\n");
        exit(EXIT_FAILURE);
    }

    input_file = argv[optind];

    if (remote_node == 0) {
        fprintf(stderr, "Remote node-id is not specified. Use -r <remote node-id>\n");
        exit(EXIT_FAILURE);
    }

    // Open output file
    outfile = fopen(output_file, "wb");
    if (outfile == NULL)
    {
        perror("fopen");
        exit(EXIT_FAILURE);
    }

    // Initialize encoder
    struct c63_common *cm = init_c63_enc(width, height);
    cm->e_ctx.fp = outfile;

    if (limit_numframes)
    {
        printf("Limited to %d frames.\n", limit_numframes);
    }
    cm->curframe = create_frame(cm, NULL);  // Create with NULL for a placeholder
    cm->refframe = create_frame(cm, NULL); 
    // Open input file
    FILE *infile = fopen(input_file, "rb");
    if (infile == NULL)
    {
        perror("fopen");
        exit(EXIT_FAILURE); 
    }

    // Initialize SISCI
    SCIInitialize(NO_FLAGS, &error);
    if (error != SCI_ERR_OK) {
        fprintf(stderr, "SCIInitialize failed: %s\n", SCIGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Set up SISCI resources
    sci_desc_t sd;
    sci_local_segment_t localSegment;
    sci_remote_segment_t remoteSegment;
    sci_map_t localMap, remoteMap;
    sci_dma_queue_t dmaQueue;
    volatile struct client_segment *client_segment;
    volatile struct server_segment *server_segment;

    // Open virtual device
    SCIOpen(&sd, NO_FLAGS, &error);
    if (error != SCI_ERR_OK) {
        fprintf(stderr, "SCIOpen failed - Error code 0x%x\n", error);
        SCITerminate();
        exit(EXIT_FAILURE);
    }
    
    // Create local segment
    SCICreateSegment(sd,
                     &localSegment,
                     SEGMENT_CLIENT,
                     sizeof(struct client_segment),
                     NO_CALLBACK,
                     NULL,
                     NO_FLAGS,
                     &error);
    if (error != SCI_ERR_OK) {
        fprintf(stderr, "SCICreateSegment failed - Error code 0x%x\n", error);
        SCIClose(sd, NO_FLAGS, &error);
        SCITerminate();
        exit(EXIT_FAILURE);
    }
    
    // Prepare segment
    SCIPrepareSegment(localSegment, localAdapterNo, NO_FLAGS, &error);
    if (error != SCI_ERR_OK) {
        fprintf(stderr, "SCIPrepareSegment failed - Error code 0x%x\n", error);
        SCIRemoveSegment(localSegment, NO_FLAGS, &error);
        SCIClose(sd, NO_FLAGS, &error);
        SCITerminate();
        exit(EXIT_FAILURE);
    }
    
    // Create DMA queue
    SCICreateDMAQueue(sd, &dmaQueue, localAdapterNo, 1, NO_FLAGS, &error);
    if (error != SCI_ERR_OK) {
        fprintf(stderr, "SCICreateDMAQueue failed - Error code 0x%x\n", error);
        SCIRemoveSegment(localSegment, NO_FLAGS, &error);
        SCIClose(sd, NO_FLAGS, &error);
        SCITerminate();
        exit(EXIT_FAILURE);
    }
    
    // Map local segment
    client_segment = (volatile struct client_segment *)SCIMapLocalSegment(
        localSegment, 
        &localMap, 
        0, 
        sizeof(struct client_segment), 
        NULL, 
        NO_FLAGS, 
        &error);
    
    if (error != SCI_ERR_OK) {
        fprintf(stderr, "SCIMapLocalSegment failed - Error code 0x%x\n", error);
        SCIRemoveDMAQueue(dmaQueue, NO_FLAGS, &error);
        SCIRemoveSegment(localSegment, NO_FLAGS, &error);
        SCIClose(sd, NO_FLAGS, &error);
        SCITerminate();
        exit(EXIT_FAILURE);
    }
    
    // Initialize control packet
    client_segment->packet.cmd = CMD_INVALID;
    
    // Make segment available
    SCISetSegmentAvailable(localSegment, localAdapterNo, NO_FLAGS, &error);
    if (error != SCI_ERR_OK) {
        fprintf(stderr, "SCISetSegmentAvailable failed - Error code 0x%x\n", error);
        SCIUnmapSegment(localMap, NO_FLAGS, &error);
        SCIRemoveDMAQueue(dmaQueue, NO_FLAGS, &error);
        SCIRemoveSegment(localSegment, NO_FLAGS, &error);
        SCIClose(sd, NO_FLAGS, &error);
        SCITerminate();
        exit(EXIT_FAILURE);
    }
    
    printf("Client: Connecting to server segment...\n");
    
    // Connect to server segment
    do {
        SCIConnectSegment(sd,
                          &remoteSegment,
                          remote_node,
                          SEGMENT_SERVER,
                          localAdapterNo,
                          NO_CALLBACK,
                          NULL,
                          SCI_INFINITE_TIMEOUT,
                          NO_FLAGS,
                          &error);
    } while (error != SCI_ERR_OK);
    
    printf("Client: Connected to server segment\n");
    
    // Map remote segment
    server_segment = (volatile struct server_segment *)SCIMapRemoteSegment(
        remoteSegment, 
        &remoteMap, 
        0,
        sizeof(struct server_segment),
        NULL, 
        NO_FLAGS, 
        &error);
    
    if (error != SCI_ERR_OK) {
        fprintf(stderr, "SCIMapRemoteSegment failed - Error code 0x%x\n", error);
        printf("Segment handle: %p\n", (void*)remoteSegment);
        printf("Map handle: %p\n", (void*)remoteMap);
        SCIUnmapSegment(remoteMap, NO_FLAGS, &error);
        SCIDisconnectSegment(remoteSegment, NO_FLAGS, &error);
        SCISetSegmentUnavailable(localSegment, localAdapterNo, NO_FLAGS, &error);
        SCIUnmapSegment(localMap, NO_FLAGS, &error);
        SCIRemoveDMAQueue(dmaQueue, NO_FLAGS, &error);
        SCIRemoveSegment(localSegment, NO_FLAGS, &error);
        SCIClose(sd, NO_FLAGS, &error);
        SCITerminate();
        exit(EXIT_FAILURE);
    }
    
    // Enter main processing loop
    main_client_loop(cm, infile, limit_numframes, client_segment, server_segment, 
                     dmaQueue, localSegment, remoteSegment);
    
    // Clean up resources
    destroy_frame(cm->refframe);
    fclose(outfile);
    fclose(infile);
    free_c63_enc(cm);
    
    SCIUnmapSegment(remoteMap, NO_FLAGS, &error);
    SCIDisconnectSegment(remoteSegment, NO_FLAGS, &error);
    SCISetSegmentUnavailable(localSegment, localAdapterNo, NO_FLAGS, &error);
    SCIUnmapSegment(localMap, NO_FLAGS, &error);
    SCIRemoveDMAQueue(dmaQueue, NO_FLAGS, &error);
    SCIRemoveSegment(localSegment, NO_FLAGS, &error);
    SCIClose(sd, NO_FLAGS, &error);
    SCITerminate();
    
    return 0;
}